#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/imgproc/imgproc.hpp>
#include<hip/hip_runtime.h>

using std::cout;
using std::endl;

__global__ void Kernel( unsigned char* inputA, unsigned char* inputB, 
 
	unsigned char* outputF, int width, int height,	 int grayWidthStep)
{
	
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//unsigned char outputA[640*480];
	
	
	if((xIndex>0)  && (yIndex>0) && (xIndex<width) && (yIndex<height))
	{
		
		const int gray_tid  = yIndex * grayWidthStep + xIndex;

		outputF[gray_tid]=inputB[gray_tid]-inputA[gray_tid];
		
		__syncthreads();

		
		
	}
}


void ImageGrayConverter(const cv::Mat& FrameA, const cv::Mat& FrameB, cv::Mat& FrameFinal)
{
	
	//const int colorBytes = FrameA.step * FrameA.rows;
	const int grayBytes = FrameFinal.step * FrameFinal.rows;

	unsigned char *D_FrameA, *D_FrameB, *D_FrameFinal;
	
	hipMalloc<unsigned char>(&D_FrameA,grayBytes);
	hipMalloc<unsigned char>(&D_FrameB,grayBytes);
		
	hipMalloc<unsigned char>(&D_FrameFinal,grayBytes);
	
	hipMemcpy(D_FrameA, FrameA.ptr(),grayBytes,hipMemcpyHostToDevice);
	hipMemcpy(D_FrameB, FrameB.ptr(),grayBytes,hipMemcpyHostToDevice);
		
	const dim3 block(32,32);
	
	const dim3 grid((FrameA.cols + block.x - 1)/block.x, (FrameA.rows + block.y - 1)/block.y);
	
	Kernel<<<grid,block>>>(D_FrameA, D_FrameB, D_FrameFinal, FrameA.cols, FrameA.rows, 

	FrameFinal.step);
	
	hipDeviceSynchronize();	
	
	hipMemcpy(FrameFinal.ptr(),D_FrameFinal,grayBytes,hipMemcpyDeviceToHost);
	
	hipFree(D_FrameA);
	hipFree(D_FrameB);
			
	hipFree(D_FrameFinal);
}

using namespace std;
using namespace cv;

int main(int argc, char** argv)
{
	Mat InputA;	
	Mat InputB;
	Mat Output;
	Mat Gray;

	cv::VideoCapture cap;
	cap.open(string(argv[1]));	

	//Output = Mat::zeros(Input.rows,Input.cols,CV_8UC1);
	//Mat Gray = Mat::zeros(Input.rows,Input.cols,CV_8UC1);

	//cvtColor(Input,Gray,CV_BGR2GRAY);
	
	        						

	while(1)
	{	
		cap>>InputA;
		cap>>InputB;	
			
                cvtColor(InputA,InputA,CV_BGR2GRAY);
		cvtColor(InputB,InputB,CV_BGR2GRAY);

		//Output=InputB-InputA;
		//cvtColor(Output,Gray,CV_BGR2GRAY);

		cv::Mat FrameFinal(InputA.rows,InputA.cols,CV_8U);

		ImageGrayConverter(InputA,InputB,FrameFinal);		
		cv::threshold(FrameFinal,FrameFinal, 225, 255, cv::THRESH_BINARY);		
		cv::imshow("Input",InputA);		
		cv::imshow("Output",FrameFinal);
		//cv::imshow("Output_CPU",Gray);		
		if(cv::waitKey(33)>=0) break;
	
	}	
	return 0;
}
