#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/imgproc/imgproc.hpp>
#include<hip/hip_runtime.h>
#include<time.h>

using std::cout;
using std::endl;

__global__ void Kernel(unsigned char* FrameA,unsigned char* FrameB,unsigned char* Frame,unsigned char* Gray,unsigned char* Bin,unsigned char* Ero,unsigned char* Dil,unsigned char* ExA,unsigned char* ExB,unsigned char* ExC,unsigned char* ExD,unsigned char* ExE,unsigned char* FrameF,int width,int height,int colorWidthStep, int grayWidthStep)
{
	
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	
	if((xIndex>1)  && (yIndex>1) && (xIndex<width-1) && (yIndex<height-1))
	{
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		const int gray_tid  = yIndex * grayWidthStep + xIndex;

		Frame[color_tid]=FrameB[color_tid]-FrameA[color_tid];	
		Frame[color_tid+1]=FrameB[color_tid+1]-FrameA[color_tid+1];	
		Frame[color_tid+2]=FrameB[color_tid+2]-FrameA[color_tid+2];	
	
		__syncthreads();

		const unsigned char blue	= Frame[color_tid];
		const unsigned char green	= Frame[color_tid + 1];
		const unsigned char red		= Frame[color_tid + 2];

		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

		Gray[gray_tid] = static_cast<unsigned char>(gray);	

		__syncthreads();

		if(Gray[gray_tid]>220)
			Bin[gray_tid]=255;
		else
			Bin[gray_tid]=0;

		__syncthreads();	

		const int tidA = (yIndex) * grayWidthStep + (xIndex); 		// x , y
		const int tidB = (yIndex-1) * grayWidthStep + (xIndex); 	// x , y-1
		const int tidC = (yIndex+1) * grayWidthStep + (xIndex); 	// x , y+1
		const int tidD = (yIndex) * grayWidthStep + (xIndex-1);		// x-1 , y
		const int tidE = (yIndex) * grayWidthStep + (xIndex+1);		// x+1 , y	
		const int tidF = (yIndex-1) * grayWidthStep + (xIndex-1);	// x-1 , y-1
		const int tidG = (yIndex-1) * grayWidthStep + (xIndex+1);	// x+1 , y-1
		const int tidH = (yIndex+1) * grayWidthStep + (xIndex-1);	// x-1 , y+1	
		const int tidI = (yIndex+1) * grayWidthStep + (xIndex+1);	// x+1 , y+1		
		
		const int tidJ = (yIndex) * grayWidthStep + (xIndex-2);		// x-2 , y
		const int tidK = (yIndex) * grayWidthStep + (xIndex+2);		// x+2 , y	
		const int tidL = (yIndex-1) * grayWidthStep + (xIndex-2);	// x-2 , y-1
		const int tidM = (yIndex-1) * grayWidthStep + (xIndex+2);	// x+2 , y-1
		const int tidN = (yIndex+1) * grayWidthStep + (xIndex-2);	// x-1 , y+1	
		const int tidO = (yIndex+1) * grayWidthStep + (xIndex+2);	// x+1 , y+1		
				
		const int tidP = (yIndex-2) * grayWidthStep + (xIndex-1);	// x-1 , y-1
		const int tidQ = (yIndex-2) * grayWidthStep + (xIndex);		// x   , y-2	
		const int tidR = (yIndex-2) * grayWidthStep + (xIndex+1);	// x+1 , y-2
		const int tidS = (yIndex+2) * grayWidthStep + (xIndex-1);	// x+2 , y-1
		const int tidT = (yIndex+2) * grayWidthStep + (xIndex);		// x-1 , y+1	
		const int tidU = (yIndex+2) * grayWidthStep + (xIndex+1);	// x+1 , y+1		
						



/**/		
	
		if((Bin[tidA]>100)&&(Bin[tidB]>100)&&(Bin[tidD]>100)&&(Bin[tidE]>100)&&(Bin[tidG]>100)&&(Bin[tidF]>100))
		Ero[gray_tid]=255;
		else
		Ero[gray_tid]=0;
		
		__syncthreads();

		if((Ero[tidA]>100)&&(Ero[tidB]>100)&&(Ero[tidD]>100)&&(Ero[tidE]>100)&&(Ero[tidG]>100)&&(Ero[tidF]>100))
		Dil[gray_tid]=255;
		else
		Dil[gray_tid]=0;
		
		__syncthreads();

		if((Dil[tidA]>100)&&(Dil[tidB]>100)&&(Dil[tidD]>100)&&(Dil[tidE]>100)&&(Dil[tidG]>100)&&(Dil[tidF]>100))
		ExA[gray_tid]=255;
		else
		ExA[gray_tid]=0;

		__syncthreads();

		if((ExA[tidA]>100)||(ExA[tidB]>100)||(ExA[tidC]>100)||(ExA[tidD]>100)||(ExA[tidE]>100)||
		(ExA[tidF]>100)||(ExA[tidG]>100)||(ExA[tidH]>100)||(ExA[tidI]>100)||(ExA[tidJ]>100)||(ExA[tidK]>100)
		||(ExA[tidL]>100)||(ExA[tidM]>100)||(ExA[tidN]>100)||(ExA[tidO]>100))
		ExB[gray_tid]=255;
		else
		ExB[gray_tid]=0;

		__syncthreads();

		if((ExB[tidA]>100)||(ExB[tidB]>100)||(ExB[tidC]>100)||(ExB[tidD]>100)||(ExB[tidE]>100)||
		(ExB[tidF]>100)||(ExB[tidG]>100)||(ExB[tidH]>100)||(ExB[tidI]>100)||(ExB[tidJ]>100)||(ExB[tidK]>100)
		||(ExB[tidL]>100)||(ExB[tidM]>100)||(ExB[tidN]>100)||(ExB[tidO]>100))
		ExC[gray_tid]=255;
		else
		ExC[gray_tid]=0;

		__syncthreads();

		if((ExC[tidA]>100)||(ExC[tidB]>100)||(ExC[tidC]>100)||(ExC[tidD]>100)||(ExC[tidE]>100)||
		(ExC[tidF]>100)||(ExC[tidG]>100)||(ExC[tidH]>100)||(ExC[tidI]>100)||(ExC[tidJ]>100)||(ExC[tidK]>100)
		||(ExC[tidL]>100)||(ExC[tidM]>100)||(ExC[tidN]>100)||(ExC[tidO]>100))
		ExD[gray_tid]=255;
		else
		ExD[gray_tid]=0;

		__syncthreads();

		if((ExD[tidA]>100)||(ExD[tidB]>100)||(ExD[tidC]>100)||(ExD[tidD]>100)||(ExD[tidE]>100)||
		(ExD[tidF]>100)||(ExD[tidG]>100)||(ExD[tidH]>100)||(ExD[tidI]>100)||(ExD[tidP]>100)||(ExD[tidQ]>100)
		||(ExD[tidR]>100)||(ExD[tidS]>100)||(ExD[tidT]>100)||(ExD[tidU]>100))
		ExE[gray_tid]=255;
		else
		ExE[gray_tid]=0;

		__syncthreads();

		if((ExE[tidA]>100)||(ExE[tidB]>100)||(ExE[tidC]>100)||(ExE[tidD]>100)||(ExE[tidE]>100)||
		(ExE[tidF]>100)||(ExE[tidG]>100)||(ExE[tidH]>100)||(ExE[tidI]>100)||(ExE[tidP]>100)||(ExE[tidQ]>100)
		||(ExE[tidR]>100)||(ExE[tidS]>100)||(ExE[tidT]>100)||(ExE[tidU]>100))
		FrameF[gray_tid]=255;
		else
		FrameF[gray_tid]=0;

		

					
		
	}
}


void ImageGrayConverter(const cv::Mat& FrameA, const cv::Mat& FrameB, cv::Mat& FrameFinal)
{
	
	const int colorBytes = FrameA.step * FrameA.rows;
	const int grayBytes = FrameFinal.step * FrameFinal.rows;

	unsigned char *D_FrameA, *D_FrameB, *D_Frame;

	unsigned char *D_Gray, *D_Bin, *D_Ero, *D_Dil, *D_ExA, *D_ExB, *D_ExC,*D_ExD,*D_ExE, *D_FrameFinal;
	
	hipMalloc<unsigned char>(&D_FrameA,colorBytes);
	hipMalloc<unsigned char>(&D_FrameB,colorBytes);
	hipMalloc<unsigned char>(&D_Frame,colorBytes);

	hipMalloc<unsigned char>(&D_Gray,grayBytes);
	hipMalloc<unsigned char>(&D_Bin,grayBytes);
	hipMalloc<unsigned char>(&D_Ero,grayBytes);
	hipMalloc<unsigned char>(&D_Dil,grayBytes);
	hipMalloc<unsigned char>(&D_ExA,grayBytes);
	hipMalloc<unsigned char>(&D_ExB,grayBytes);
	hipMalloc<unsigned char>(&D_ExC,grayBytes);
	hipMalloc<unsigned char>(&D_ExD,grayBytes);
	hipMalloc<unsigned char>(&D_ExE,grayBytes);
		
	hipMalloc<unsigned char>(&D_FrameFinal,grayBytes);
	
	hipMemcpy(D_FrameA, FrameA.ptr(),colorBytes,hipMemcpyHostToDevice);
	hipMemcpy(D_FrameB, FrameB.ptr(),colorBytes,hipMemcpyHostToDevice);
		
	const dim3 block(32,32);
	
	const dim3 grid((FrameA.cols + block.x - 1)/block.x, (FrameA.rows + block.y - 1)/block.y);
	
	Kernel<<<grid,block>>>(D_FrameA,D_FrameB,D_Frame,D_Gray,D_Bin,D_Ero,D_Dil,D_ExA,D_ExB,D_ExC,D_ExD,D_ExE,D_FrameFinal,
				FrameA.cols,FrameA.rows,FrameA.step,FrameFinal.step);
	
	hipDeviceSynchronize();	
	
	hipMemcpy(FrameFinal.ptr(),D_FrameFinal,grayBytes,hipMemcpyDeviceToHost);
	
	hipFree(D_FrameA);
	hipFree(D_FrameB);
	hipFree(D_Frame);
	hipFree(D_Gray);
	hipFree(D_Bin);
	hipFree(D_Ero);
	hipFree(D_Dil);
	hipFree(D_ExA);
	hipFree(D_ExB);
	hipFree(D_ExC);
	hipFree(D_ExD);
	hipFree(D_ExE);			
	hipFree(D_FrameFinal);
}

using namespace std;
using namespace cv;

int main(int argc, char** argv)
{
	Mat InputA;	
	Mat InputB;

	cv::VideoCapture cap;
	cap.open(0); 

	clock_t Time_Start, Time_End, Time_Difference; // Clock used to measure time for CPU 
	double Time;						

	while(1)
	{	
		Time_Start=clock();

		cap>>InputA;
		cap>>InputB;				
     
		cv::Mat FrameFinal(InputA.rows,InputA.cols,CV_8U);

		ImageGrayConverter(InputA,InputB,FrameFinal);

		Time_End=clock();
		Time_Difference=Time_End-Time_Start;
		Time=Time_Difference/(double)CLOCKS_PER_SEC ;
		printf ("GPU Frame Rate = %f FPS\n",1/Time);
			
		RNG rng(12345);
		vector<vector<Point> >contours;
  		vector<Vec4i>hierarchy;

		findContours(FrameFinal,contours,hierarchy,CV_RETR_TREE,CV_CHAIN_APPROX_SIMPLE,Point(0, 0));

		vector<vector<Point> > contours_poly(contours.size());
 		vector<Rect>boundRect(contours.size());
		vector<Point2f>center(contours.size());
 		vector<float>radius(contours.size());

  		for(int i=0;i<contours.size();i++)
			{ 
				approxPolyDP(Mat(contours[i]),contours_poly[i],3,true);
				boundRect[i]=boundingRect(Mat(contours_poly[i]));
				minEnclosingCircle((Mat)contours_poly[i],center[i],radius[i]);
			}


		for(int i=0;i<contours.size();i++)
			{
				Scalar color = Scalar(0,0,255);
				rectangle(InputA,boundRect[i].tl(),boundRect[i].br(),color,2,8,0);
			}
			
		cv::imshow("GPU Accelerated Tracking",InputA);	
		cv::imshow("Prcessed Frame",FrameFinal);		
		
		if(cv::waitKey(33)>=0) break;	
	}	
	return 0;
}
