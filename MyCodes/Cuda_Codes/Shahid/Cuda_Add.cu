
#include <hip/hip_runtime.h>
#include<stdio.h>

#define Width	32
#define Height	32

__global__ void Kernel(int *a, int *b, int *c, int *d)
{
	int x= blockIdx.x * blockDim.x + threadIdx.x;
	int y= blockIdx.y * blockDim.y + threadIdx.y;

	int id= y*Width+x;

	d[id]= (a[id] + b[id]) - (b[id] + c[id]);

}

int main(void)
{
	int *h_a, *h_b, *h_c, *h_d;
	int *d_a, *d_b, *d_c, *d_d;

	int Size= Width*Height*sizeof(int);

	int i,j;

	h_a= (int *)malloc(Size);
	h_b= (int *)malloc(Size);
	h_c= (int *)malloc(Size);
	h_d= (int *)malloc(Size);

	hipMalloc(&d_a,Size);
	hipMalloc(&d_b,Size);
	hipMalloc(&d_c,Size);
	hipMalloc(&d_d,Size);

	for(i=0;i<Width;i++)
		for(j=0;j<Height;j++)
			{
				h_a[i*Width+j]=32;
				h_b[i*Width+j]=32;
				h_c[i*Width+j]=16;
				h_d[i*Width+j]=0;
			}

	hipMemcpy(d_a, h_a, Size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, Size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, Size, hipMemcpyHostToDevice);

	dim3 Blocks(32,32);
	dim3 Grid(Width/32,Height/32);

	Kernel<<<Grid,Blocks>>>(d_a,d_b,d_c,d_d);

	hipMemcpy(h_d, d_d, Size, hipMemcpyDeviceToHost);

	for(i=0;i<Width;i++)
		{
		for(j=0;j<Height;j++)
			{
				printf("%d ",h_d[i*Width+j]);
			}
		printf("\n");
		}

	free(h_a);
	free(h_b);
	free(h_c);
	free(h_d);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_d);


return 0;
}
