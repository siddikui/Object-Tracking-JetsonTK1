
#include <hip/hip_runtime.h>
#include<stdio.h>

#define W 16
#define H 16
#define Mask 3

__global__ void Conv(int *input, int *output, int *mask)
{

	int x= blockIdx.x * Mask + threadIdx.x; // Thread Column Index.
	int y= blockIdx.y * Mask + threadIdx.y; // Thread Row Index.
	
	
	int Sum=0;	

	
	for(int i=-1;i<=1;i++)
		for(int j=-1;j<=1;j++)
			Sum+= input[(y+j)*W+(x+i)]*mask[(j+1)*Mask+(i+1)];

	output[y*W+x] = Sum; // Writes result to Output Image pixel.

}

int main(void)
{

	int *h_input, *h_output,*h_mask;
	int *d_input, *d_output, *d_mask;

	

	int SIZE= W*H*sizeof(int);
	
	h_input=(int*)malloc(SIZE);
	h_output=(int*)malloc(SIZE);
	h_mask=(int*)malloc(Mask*Mask*sizeof(int));
	
	for(int k=0; k<Mask*Mask;k++)
	h_mask[k]=1;
	
	for(int i=0;i<W;i++){
		for(int j=0;j<H;j++)
		{
				
		h_input[i*W+j]=1;
		h_output[i*W+j]=0;
				
		}
	}
		
	hipMalloc(&d_input, SIZE);
	hipMalloc(&d_output, SIZE);
	hipMalloc(&d_mask, Mask*Mask*sizeof(int));

	hipMemcpy(d_input, h_input, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_mask, h_mask, Mask*Mask*sizeof(int), hipMemcpyHostToDevice);

	dim3 threads(W/Mask, H/Mask);
	dim3 blocks(Mask, Mask);	
	
	Conv<<<blocks, threads>>>(d_input, d_output,d_mask);

	hipMemcpy(h_output, d_output,SIZE, hipMemcpyDeviceToHost);

	int y=0;
	for(int i=0; i<W*H; i++)
	{
		if(y==W)
		{
			
		printf("\n");
		
				
		}
		printf("%d ",h_output[i]);
		y++;
	}


	free(h_input);
	free(h_output);
	free(h_mask);

	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_mask);

return 0;
}
